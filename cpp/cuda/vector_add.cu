#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 100000000

__global__ void vectorAdd(float* out, float* a, float* b, int n)
{
  int i = threadIdx.x;
  out[i] = a[i] + b[i];
}

int main()
{
  float *a, *b, *out;
  float *d_a, *d_b, *d_out;

  // Allocate memory
  a = (float*)malloc(sizeof(float) * N);
  b = (float*)malloc(sizeof(float) * N);
  out = (float*)malloc(sizeof(float) * N);

  // Initialize array
  for (int i = 0; i < N; i++)
    {
      a[i] = 1.0f;
      b[i] = 2.0f;
    }

  // Allocate device memory
  hipMalloc((void**)&d_a, sizeof(float) * N);
  hipMalloc((void**)&d_b, sizeof(float) * N);
  hipMalloc((void**)&d_out, sizeof(float) * N);

  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  // Executing kernel
  vectorAdd<<<1,1>>>(d_out, d_a, d_b, N);

  // Transfer data back to host memory
  hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

  printf("%f\n", out[0]);

  // Deallocate device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  // Deallocate host memory
  free(a);
  free(b);
  free(out);
}
