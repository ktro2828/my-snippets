#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


constexpr int N = 10;

__global__ void matrixAdd(float** out, float** a, float** b, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < N && j < N)
  {
    out[i][j] = a[i][j] + b[i][j];
  }
}

int main()
{
  float **a, **b, **out;
  float **d_a, **d_b, **d_out;

  // Allocate memory
  a = (float**)malloc(sizeof(float*) * N);
  b = (float**)malloc(sizeof(float*) * N);
  out = (float**)malloc(sizeof(float*) * N);

  // Initialize array
  for (int i = 0; i < N; i++)
    {
      // Allocate memory for each row of the matrix.
      a[i] = (float*)malloc(sizeof(float) * N);
      b[i] = (float*)malloc(sizeof(float) * N);
      out[i] = (float*)malloc(sizeof(float) * N);
      for (int j = 0; j < N; j++)
      {
        a[i][j] = 1.0f;
        b[i][j] = 2.0f;
      }
    }

  // Allocate device memory
  hipMalloc((void**)&d_a, sizeof(float*) * N);
  hipMalloc((void**)&d_b, sizeof(float*) * N);
  hipMalloc((void**)&d_out, sizeof(float*) * N);
  for (int i = 0; i < N; i++)
  {
    // Allocate memory  on device
    hipMalloc((void**)&(d_a[i]), sizeof(float) * N);
    hipMalloc((void**)&(d_b[i]), sizeof(float) * N);
    hipMalloc((void**)&(d_out[i]), sizeof(float) * N);
  }

  // Transfer data from host to device
  hipMemcpy(d_a, a, sizeof(float*) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float*) * N, hipMemcpyHostToDevice);
  for (int i = 0; i < N; i++)
  {
    hipMemcpy(d_a[i], a[i], sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b[i], b[i], sizeof(float) * N, hipMemcpyHostToDevice);
  }

  // Executing kernel
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
  matrixAdd<<<numBlocks, threadsPerBlock>>>(d_out, d_a, d_b, N);

  // Transfer data back to host memory
  hipMemcpy(out, d_out, sizeof(float*) * N, hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++)
  {
    hipMemcpy(out[i], d_out[i], sizeof(float) * N, hipMemcpyDeviceToHost);
  }

  printf("%f\n", out[0][0]);

  // // Deallocate device memory
  for (int i = 0; i < N; i++)
  {
    hipFree(d_a[i]);
    hipFree(d_b[i]);
    hipFree(d_out[i]);
    free(a[i]);
    free(b[i]);
    free(out[i]);
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  // Deallocate host memory
  free(a);
  free(b);
  free(out);
}
